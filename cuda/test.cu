#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;


__global__ void mul_kernel(int *d_in, int *d_out, int scalar)
{
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[globalId] = d_in[globalId]*scalar;
}

float mul(int *d_in, int *d_out, int blockSize, int gridSize)
{
    int scalar = 3;
    dim3 grid(gridSize);
    dim3 block(blockSize);

    float totalTime;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    mul_kernel<<<grid, block>>>(d_in, d_out, scalar);
    hipEventRecord(end);

    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&totalTime, start, end);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout<<hipGetErrorString(err)<<std::endl;

    return totalTime;
}

void testMem() {
    hipError_t err;
    int blockSize = 1024, gridSize = 32768;
    int len = blockSize * gridSize;
    std::cout<<"Data size(Multiplication): "<<len<<" ("<<len* sizeof(int)/1024/1024<<"MB)"<<std::endl;

    float mulTime = 0.0;

    int *h_in, *d_in, *d_out;
    h_in = new int[len];
    for(int i = 0; i < len; i++) {
        h_in[i] = i;
    }
    checkCudaErrors(hipMalloc(&d_in,sizeof(int)*len));
    checkCudaErrors(hipMalloc(&d_out,sizeof(int)*len));
    hipMemcpy(d_in, h_in, sizeof(int)*len, hipMemcpyHostToDevice);

    for(int i = 0; i < 10; i++) {
        float tempTime = mul(d_in, d_out, blockSize, gridSize);

        //throw away the first result
        if (i != 0)     mulTime += tempTime;
    }
    mulTime /= (10- 1);

    delete[] h_in;
    checkCudaErrors(hipFree(d_out));
}

int main() {
    testMem();
    return 0;
}

