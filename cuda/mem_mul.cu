#include "hip/hip_runtime.h"
/*
 * compile: nvcc -o mem_mul -arch=sm_35 -O3 mem_mul.cu -I /usr/local/cuda/samples/common/inc/
 */
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;

#define SCALAR  (3)

__global__ void mul_kernel(int *d_in, int *d_out, int num)
{
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int globalSIze = blockDim.x * gridDim.x;

    while (globalId < num) {
        d_out[globalId] = d_in[globalId]*SCALAR;
        globalId += globalSIze;
    }
}

float mul(int *d_in, int *d_out, int num)
{
    int blockSize = 1024, gridSize = 32768;
    dim3 grid(gridSize);
    dim3 block(blockSize);

    float totalTime;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    mul_kernel<<<grid, block>>>(d_in, d_out, num);
    hipEventRecord(end);

    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&totalTime, start, end);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout<<hipGetErrorString(err)<<std::endl;

    return totalTime;
}

void test_bandwidth() {

    int len = 512 * 8192 * 100;     //1600MB
    std::cout<<"Data size(Multiplication): "<<len<<" ("<<len* sizeof(int)/1024/1024<<"MB)"<<std::endl;

    float mulTime = 0.0;

    int *h_in, *d_in, *d_out;
    h_in = new int[len];
    for(int i = 0; i < len; i++) {
        h_in[i] = i;
    }
    checkCudaErrors(hipMalloc(&d_in,sizeof(int)*len));
    checkCudaErrors(hipMalloc(&d_out,sizeof(int)*len));
    hipMemcpy(d_in, h_in, sizeof(int)*len, hipMemcpyHostToDevice);

    int experTime = 10;
    for(int i = 0; i < experTime; i++) {
        float tempTime = mul(d_in, d_out, len);
        if (i != 0)     mulTime += tempTime;
    }
    mulTime /= (experTime - 1);

    delete[] h_in;
    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));

    //both read and write
    double throughput = 2*sizeof(int)*len / mulTime / 1e6;

    std::cout<<"Time for multiplication: "<<mulTime<<" ms."<<'\t'
        <<"Bandwidth: "<<throughput<<" GB/s"<<std::endl;
}

int main() {
    hipSetDevice(1);
    test_bandwidth();
    return 0;
}