#include "hip/hip_runtime.h"
//
//  map.cu
//  comparison_gpu/cuda
//
//  Created by Zhuohang Lai on 01/19/16.
//  Copyright (c) 2015-2016 Zhuohang Lai. All rights reserved.
//
#include "kernels.h"

__global__ void copy_kernel(int *d_in, int *d_out, int scalar)
{
	int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[globalId] = d_in[globalId]*scalar;
}

float mul(int *d_in, int *d_out, int blockSize, int gridSize)
{
    int scalar = 3;
	dim3 grid(gridSize);
	dim3 block(blockSize);

	float totalTime;

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);
    copy_kernel <<<grid, block>>>(d_in, d_out, scalar);
	hipEventRecord(end);

    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&totalTime, start, end);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout<<hipGetErrorString(err)<<std::endl;

	return totalTime;
}


