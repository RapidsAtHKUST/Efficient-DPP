#include "hip/hip_runtime.h"
//
//  map.cu
//  gpuqp_cuda
//
//  Created by Bryan on 01/19/16.
//  Copyright (c) 2015-2016 Bryan. All rights reserved.
//
#include "kernels.h"

//mapping function 1:
__device__ Record floorOfPower2(Record a) {
	int base = 1;
	while (base < a.y) {
		base <<= 1;
	}
	a.y = (base>>1);
	return a;
}

template<typename T> 
__global__ void map_kernel(T *d_source, T *d_res, int r_len) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = gridDim.x * blockDim.x;
	
	while (threadId < r_len) {
		d_res[threadId] = floorOfPower2(d_source[threadId]);
		threadId += threadNum;
	}
}

template<typename T>
double map(T *d_source, T *d_res, int r_len, int blockSize, int gridSize) {

	dim3 grid(gridSize);
	dim3 block(blockSize);

	double totalTime = 0.0f;
	struct timeval start, end;

	gettimeofday(&start, NULL);
	map_kernel<T><<<grid, block>>>(d_source, d_res, r_len);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);

	totalTime = diffTime(end, start);

	return totalTime;
}

double mapImpl(Record *h_source, Record *h_res, int r_len, int blockSize, int gridSize) {

	double totalTime = 0.0f;
	
	Record *d_source, *d_res;
	
	//allocate for the device memory
	checkCudaErrors(hipMalloc(&d_source,sizeof(Record)*r_len));
	checkCudaErrors(hipMalloc(&d_res,sizeof(Record)*r_len));

	hipMemcpy(d_source, h_source, sizeof(Record) * r_len, hipMemcpyHostToDevice);
	totalTime = map<Record>(d_source, d_res, r_len, blockSize, gridSize);
	hipMemcpy(h_res, d_res, sizeof(Record)*r_len, hipMemcpyDeviceToHost);	
	
	checkCudaErrors(hipFree(d_res));
	checkCudaErrors(hipFree(d_source));

	return totalTime;
}





