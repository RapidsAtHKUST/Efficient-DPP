#include "hip/hip_runtime.h"
//
//  gather.cu
//  gpuqp_cuda
//
//  Created by Bryan on 01/19/16.
//  Copyright (c) 2015-2016 Bryan. All rights reserved.
//
#include "kernels.h"

__global__
void gather(const Record *d_source,
			Record *d_res,
			const int r_len,
			const int *loc)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;

	while (threadId < r_len) {
		//this is awesomely faster if loc[threadId] is stored in local register
		d_res[threadId] = d_source[loc[threadId]];

		// d_res[threadId].x = d_source[loc[threadId]].x;
		// d_res[threadId].y = d_source[loc[threadId]].y;
		threadId += threadNum;
	}
}

double gatherDevice(Record *d_source, Record *d_res, int r_len,int *d_loc, int blockSize, int gridSize) {
	dim3 grid(gridSize);
	dim3 block(blockSize);

	double totalTime = 0.0f;
	struct timeval start, end;

	gettimeofday(&start, NULL);
	gather<<<grid, block>>>(d_source, d_res, r_len, d_loc);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);

	totalTime = diffTime(end, start);

	return totalTime;
}

double gatherImpl(Record *h_source, Record *h_res, int r_len,int *h_loc, int blockSize, int gridSize) {
	Record *d_source, *d_res;
	int *d_loc;
	double totalTime = 0.0f;

	//allocate for the device memory
	checkCudaErrors(hipMalloc(&d_source,sizeof(Record)*r_len));
	checkCudaErrors(hipMalloc(&d_res,sizeof(Record)*r_len));
	checkCudaErrors(hipMalloc(&d_loc,sizeof(int)*r_len));

	hipMemcpy(d_source, h_source, sizeof(Record) * r_len, hipMemcpyHostToDevice);
	hipMemcpy(d_loc, h_loc, sizeof(int) * r_len, hipMemcpyHostToDevice);

	totalTime = gatherDevice(d_source, d_res, r_len, d_loc, blockSize, gridSize);
	
	hipMemcpy(h_res, d_res, sizeof(Record)*r_len, hipMemcpyDeviceToHost);	
	
	checkCudaErrors(hipFree(d_res));
	checkCudaErrors(hipFree(d_source));
	checkCudaErrors(hipFree(d_loc));

	return totalTime;
}