#include "hip/hip_runtime.h"
//
//  scatter.cu
//  gpuqp_cuda
//
//  Created by Bryan on 01/19/16.
//  Copyright (c) 2015-2016 Bryan. All rights reserved.
//
#include "kernels.h"

__global__
void scatter(const Record *d_source,
			Record *d_res,
			const int r_len,
			const int *loc)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;

	while (threadId < r_len) {
		d_res[loc[threadId]].x = d_source[threadId].x;
		d_res[loc[threadId]].y = d_source[threadId].y;
		threadId += threadNum;
	}
}

void scatterImpl(Record *h_source, Record *h_res, int r_len,int *h_loc, int blockSize, int gridSize, double& time) {
	
	Record *d_source, *d_res;
	int *d_loc;

	dim3 grid(gridSize);
	dim3 block(blockSize);
	
	//allocate for the device memory
	checkCudaErrors(hipMalloc(&d_source,sizeof(Record)*r_len));
	checkCudaErrors(hipMalloc(&d_res,sizeof(Record)*r_len));
	checkCudaErrors(hipMalloc(&d_loc,sizeof(int)*r_len));

	hipMemcpy(d_source, h_source, sizeof(Record) * r_len, hipMemcpyHostToDevice);
	hipMemcpy(d_loc, h_loc, sizeof(int) * r_len, hipMemcpyHostToDevice);

	struct timeval start, end;

	gettimeofday(&start, NULL);	
	hipDeviceSynchronize();
	scatter<<<grid, block>>>(d_source, d_res, r_len, d_loc);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);

	time = diffTime(end, start);
	
	hipMemcpy(h_res, d_res, sizeof(Record)*r_len, hipMemcpyDeviceToHost);	
	
	checkCudaErrors(hipFree(d_res));
	checkCudaErrors(hipFree(d_source));
	checkCudaErrors(hipFree(d_loc));
}