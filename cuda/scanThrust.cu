#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;

bool testScan_thrust(int len, float& totalTime, int isExclusive) {

    bool res = true;

    //allocate for the host memory
    int *h_in_gpu = new int[len];
    int *h_in_cpu = new int[len];

    for(int i = 0; i < len; i++) {
        h_in_gpu[i] = 1;
        h_in_cpu[i] = 1;
    }

    int *d_in, *d_out;
    checkCudaErrors(hipMalloc(&d_in,sizeof(int)*len));
    checkCudaErrors(hipMalloc(&d_out,sizeof(int)*len));

    hipMemcpy(d_in, h_in_gpu, sizeof(int) * len, hipMemcpyHostToDevice);

    thrust::device_ptr<int> g_ptr_in = thrust::device_pointer_cast(d_in);
    thrust::device_ptr<int> g_ptr_out = thrust::device_pointer_cast(d_out);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    thrust::exclusive_scan(g_ptr_in, g_ptr_in + len, g_ptr_out);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&totalTime, start, end);

    hipMemcpy(h_in_gpu, d_out, sizeof(int) * len, hipMemcpyDeviceToHost);

    for(int i = 0; i < len; i++) {
        if (h_in_gpu[i] != i) {
            res = false;
        }
    }

//    checkCudaErrors(hipFree(d_in));

    delete[] h_in_gpu;
    delete[] h_in_cpu;

    return res;
}

int main() {
    float idnElapsedTime;
    int dataSize = 16000000;
    bool res = testScan_thrust(dataSize, idnElapsedTime, 1);
    if (res)    cout<<"right"<<endl;
    else
        cout<<"wrong"<<endl;
    cout<<"data size:"<<dataSize* sizeof(int)/1024/1024<<" MB"<<endl;
    cout<<"total time:"<<idnElapsedTime<<" ms"<<endl;
    cout<<"throughput: "<<dataSize* sizeof(int)/idnElapsedTime/1e6<<" GB/s"<<endl;

    return 0;
}