#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
using namespace std;

__global__ void scan(int* lo, int length, int length_log, int *sum)
{
    int localId = threadIdx.x;
    int mask_j = (1<<(length_log-1))-1;
    int mask_k = 0;
    int temp = 1;

    int localTemp = lo[localId];
    __syncthreads();

    for(int i = 0; i < length_log; i++) {
        if (localId < (length>>1)) {            //only half of the threads execute
            int para_j = (localId >> i) & mask_j;
            int para_k = localId & mask_k;

            int j = temp - 1 + (temp<<1)*para_j;
            int k = para_k;
            lo[j+k+1] = lo[j] + lo[j+k+1];

            mask_j >>= 1;
            mask_k = (mask_k<<1)+1;
            temp <<= 1;
        }
        __syncthreads();
    }

    if (localId == length-1) *sum = lo[localId];
    lo[localId] -= localTemp;
    __syncthreads();
}

int main() {
    int bits = 10;
    int threads = 1<<bits;
    int *input = new int[threads];
    for(int i=0; i < threads;i++) input[i] = 1;

    float totalTime;

    int *d_in, *d_sum;
    hipMalloc(&d_in, sizeof(int)*threads);
    hipMalloc(&d_sum, sizeof(int));
    hipMemcpy(d_in, input, sizeof(int)*threads, hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int sum = 0;
    hipEventRecord(start);
    scan<<<1,threads>>>(d_in, threads, bits, d_sum);
    hipEventRecord(end);

    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&totalTime, start, end);
    hipMemcpy(input, d_in, sizeof(int)*threads, hipMemcpyDeviceToHost);
    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i < threads;i++) {
        cout<<input[i]<<' ';
    }
    cout<<endl<<"sum:"<<sum<<endl;
    cout<<"Time: "<<totalTime<<" ms."<<endl;
    delete[] input;
    hipFree(d_in);

}